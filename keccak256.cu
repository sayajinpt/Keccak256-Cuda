#include <cstdint>
#include <cassert>
#include <hip/hip_runtime.h>

class Keccak256 {
public:
    static constexpr int HASH_LEN = 32;

    static void getHash(const std::uint8_t msg[], std::size_t len, std::uint8_t hashResult[HASH_LEN]);

private:
    static void absorb(std::uint64_t state[5][5]);
    static std::uint64_t rotl64(std::uint64_t x, int i);
};

#define UINT64_C(c) (c ## ULL)

constexpr int Keccak256::HASH_LEN;
constexpr int NUM_ROUNDS = 24;

constexpr unsigned char ROTATION[5][5] = {
    { 0, 36,  3, 41, 18},
    { 1, 44, 10, 45,  2},
    {62,  6, 43, 15, 61},
    {28, 55, 25, 21, 56},
    {27, 20, 39,  8, 14}
};

__device__ __forceinline__ std::uint64_t Keccak256::rotl64(std::uint64_t x, int i) {
    return ((0U + x) << i) | (x >> ((64 - i) & 63));
}

__device__ __forceinline__ void Keccak256::absorb(uint64_t state[5][5]) {
    uint64_t (*a)[5] = state;
    uint8_t r = 1;  // LFSR
    for (int i = 0; i < NUM_ROUNDS; i++) {
        // Theta step
        uint64_t c[5] = {};
        for (int x = 0; x < 5; x++) {
            for (int y = 0; y < 5; y++)
                c[x] ^= a[x][y];
        }
        for (int x = 0; x < 5; x++) {
            uint64_t d = c[(x + 4) % 5] ^ rotl64(c[(x + 1) % 5], 1);
            for (int y = 0; y < 5; y++)
                a[x][y] ^= d;
        }

        // Rho and pi steps
        uint64_t b[5][5];
        for (int x = 0; x < 5; x++) {
            for (int y = 0; y < 5; y++)
                b[y][(x * 2 + y * 3) % 5] = rotl64(a[x][y], ROTATION[x][y]);
        }

        // Chi step
        for (int x = 0; x < 5; x++) {
            for (int y = 0; y < 5; y++)
                a[x][y] = b[x][y] ^ (~b[(x + 1) % 5][y] & b[(x + 2) % 5][y]);
        }

        // Iota step
        for (int j = 0; j < 7; j++) {
            a[0][0] ^= static_cast<uint64_t>(r & 1) << ((1 << j) - 1);
            r = static_cast<uint8_t>((r << 1) ^ ((r >> 7) * 0x171));
        }
    }
}

__device__ __forceinline__ void Keccak256::getHash(const uint8_t msg[], size_t len, uint8_t hashResult[Keccak256::HASH_LEN]) {
    assert((msg != nullptr || len == 0) && hashResult != nullptr);
    uint64_t state[5][5] = {};

    // XOR each message byte into the state, and absorb full blocks
    int blockOff = 0;
    const int BLOCK_SIZE = 200 - Keccak256::HASH_LEN * 2; // Define BLOCK_SIZE here
    for (size_t i = 0; i < len; i++) {
        int j = blockOff >> 3;
        state[j % 5][j / 5] ^= static_cast<uint64_t>(msg[i]) << ((blockOff & 7) << 3);
        blockOff++;
        if (blockOff == BLOCK_SIZE) {
            absorb(state);
            blockOff = 0;
        }
    }

    // Final block and padding
    {
        int i = blockOff >> 3;
        state[i % 5][i / 5] ^= UINT64_C(0x01) << ((blockOff & 7) << 3);
        blockOff = BLOCK_SIZE - 1;
        int j = blockOff >> 3;
        state[j % 5][j / 5] ^= UINT64_C(0x80) << ((blockOff & 7) << 3);
        absorb(state);
    }

    // Uint64 array to bytes in little endian
    for (int i = 0; i < Keccak256::HASH_LEN; i++) {
        int j = i >> 3;
        hashResult[i] = static_cast<uint8_t>(state[j % 5][j / 5] >> ((i & 7) << 3));
    }
}
